#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "solve.h"

/*number of threads in a block*/
static const Int nThreads = 128;

/*Matrix vector multiply*/
template <class T>
__global__
void cudaMul(const Int* const rows,
			 const Int* const cols,
			 const Scalar* const an,
			 const Int N,
			 const T* const x, 
			 T* y
			 ) {
	Int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)  {
		const Int start = rows[i];
		const Int end = rows[i + 1];
		T res = an[start] * x[cols[start]];

		for (Int j = start + 1; j < end; j++)
			res -= an[j] * x[cols[j]];
		y[i] = res;
	}
}
/*jacobi solver*/
template<class T>
__global__
void cudaJacobi(const Int* const rows,
				 const Int* const cols,
				 const Scalar* const an,
				 const T* const cF,
				 T* const cF1,
				 const T* const Su,
				 T* r,
				 const Int N, 
				 Scalar omega
				 ) {
	Int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)  {
		const Int start = rows[i];
		const Int end = rows[i + 1];
		T res = Su[i], val = cF[i];

		for (Int j = start + 1; j < end; j++)
			res += an[j] * cF[cols[j]];
		res /= an[start];

		r[i] = -val;
		val *= (1 - omega);
		val += res * (omega);
		r[i] += val;
		cF1[i] = val;
	}
}
/*Taxpy*/
template<class T,class T1>
__global__
void cudaTaxpy(const Int N,
			   const T1 alpha,
			   const T* const x,
			   const T* const y,
			   T* const z
		  	   ) {
	Int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)  {
		T temp;
		temp = x[i];
		temp *= alpha;
		temp += y[i];
		z[i] = temp;
	}
}
/*Txmy*/
template<class T,class T1>
__global__
void cudaTxmy(const Int N,
			  const T* const x,
			  const T1* const y,
			  T* const z
		  	  ) {
	Int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)  {
		T temp;
		temp = x[i];
		temp *= y[i];
		z[i] = temp;
	}
}
/*Tdot*/
template <class T>
__global__ 
void Tdot(const T* const a, 
		  const T* const b, 
		  T* const c, 
		  const Int N
		  ) {
    __shared__ T cache[nThreads];
    Int tid = threadIdx.x + blockIdx.x * blockDim.x;
    Int cacheIndex = threadIdx.x;

    T   temp = T(0),val;
    while (tid < N) {
		val = a[tid];
		val *= b[tid];
        temp += val;
        tid += blockDim.x * gridDim.x;
    }  
    cache[cacheIndex] = temp;
    
    __syncthreads();

    Int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}
template<class T>
__host__ 
T cudaTdot(T* x,
		   T* y,
		   T* d_sum,
		   T* sum,
		   const Int nBlocks32,
		   const Int N
		   ) {
    Tdot <<< nBlocks32, nThreads >>> (x,y,d_sum,N);
	hipMemcpy(sum,d_sum,nBlocks32 * sizeof(T),hipMemcpyDeviceToHost);
	T c = T(0);
    for (Int i = 0; i < nBlocks32; i++)
        c += sum[i];
	return c;
}
/***********************************************
 * Template class to solve equations on GPU
 *		Solver must do many iterations to compensate
 *		for the latency caused by copying matrix
 *		from host to device.
 ***********************************************/
template<class T>
__host__
void SolveT(const MeshMatrix<T>& M) {
	const Int N = Mesh::gBCellsStart;
	const Int Nall = M.ap.size();
	const Int nBlocks = (N + nThreads - 1) / nThreads;
	const Int nBlocks32 = ((nBlocks > 32) ? 32 : nBlocks);

	//info
	if(M.flags & M.SYMMETRIC)
		MP::printH("Symmetric  : ");
	else
		MP::printH("Asymmetric : ");
	if(Controls::Solver == Controls::SOR)
		MP::print("SOR :");
	else
		MP::print("PCG :");

	/*******************************
	 *  variables on host & device
	 *******************************/
	Int*   d_rows;
	Int*   d_cols;
	Scalar*  d_an;
	Scalar*  d_anT;
	Scalar*  d_pC;
	T*       d_cF;
	T*       d_Su;
	//PCG
	T*       d_r,*d_r1;
	T*       d_p,*d_p1,*d_AP,*d_AP1;
	T        alpha,beta,o_rr,oo_rr;
	T        local_res[2];
	//reduction
	T*       sum,*d_sum;

	/*********************************
	 * allocate memory on device
	 ********************************/
	{
		CSRMatrix<T> A(M);	
		hipMalloc((void**) &d_rows,A.rows.size() * sizeof(Int));
		hipMalloc((void**) &d_cols,A.cols.size() * sizeof(Int));
		hipMalloc((void**) &d_an,  A.an.size() * sizeof(Scalar));
		hipMalloc((void**) &d_cF,  Nall * sizeof(T));
		hipMalloc((void**) &d_Su,  Nall * sizeof(T));

		hipMemcpy(d_rows ,&A.rows[0] ,A.rows.size() * sizeof(Int),  hipMemcpyHostToDevice);
		hipMemcpy(d_cols ,&A.cols[0] ,A.cols.size() * sizeof(Int),  hipMemcpyHostToDevice);
		hipMemcpy(d_an   ,&A.an[0]   ,A.an.size() * sizeof(Scalar), hipMemcpyHostToDevice);
		hipMemcpy(d_cF   ,&A.cF[0]   ,Nall *   sizeof(T),    hipMemcpyHostToDevice);
		hipMemcpy(d_Su   ,&A.Su[0]   ,Nall *   sizeof(T),    hipMemcpyHostToDevice);

		hipMalloc((void**) &d_r, Nall * sizeof(T));
		hipMalloc((void**) &d_sum, nBlocks32 * sizeof(T));
		sum = (T*) malloc(nBlocks32 * sizeof(T));

		if(Controls::Solver == Controls::SOR) {
			hipMalloc((void**) &d_AP,Nall * sizeof(T));
			hipMemcpy( d_AP,d_cF,Nall * sizeof(T),hipMemcpyDeviceToDevice);
		} else if(Controls::Solver == Controls::PCG) {
			hipMalloc((void**) &d_p,   Nall * sizeof(T));
			hipMalloc((void**) &d_AP,  Nall * sizeof(T));
			{
				ScalarCellField pC = 1./M.ap;
				hipMalloc((void**) &d_pC,N * sizeof(Scalar));
				hipMemcpy(d_pC,&pC[0],N * sizeof(Scalar),hipMemcpyHostToDevice);
			}
			if(!(M.flags & M.SYMMETRIC)) {
				hipMalloc((void**) &d_r1,   Nall * sizeof(T));
				hipMalloc((void**) &d_p1,   Nall * sizeof(T));
				hipMalloc((void**) &d_AP1,  Nall * sizeof(T));
				hipMalloc((void**) &d_anT,A.anT.size() * sizeof(Scalar));
				hipMemcpy(d_anT,&A.anT[0],A.anT.size() * sizeof(Scalar), hipMemcpyHostToDevice);
			}
		}
	}

	/*CG*/
	if(Controls::Solver == Controls::PCG) {
		hipMemset(d_r,0,Nall * sizeof(T));
		hipMemset(d_p,0,Nall * sizeof(T));
		cudaMul   <<< nBlocks, nThreads >>> (d_rows,d_cols,d_an,N,d_cF,d_AP);
		cudaTaxpy <<< nBlocks, nThreads >>> (N,Scalar(-1),d_AP,d_Su,d_r);
		cudaTxmy  <<< nBlocks, nThreads >>> (N,d_r,d_pC,d_p);
		o_rr = cudaTdot(d_r,d_p,d_sum,sum,nBlocks32,N);
	}
	/*BiCG*/
	if(!(M.flags & M.SYMMETRIC) && (Controls::Solver == Controls::PCG)) {
		hipMemcpy(d_r1,d_r,Nall * sizeof(T), hipMemcpyDeviceToDevice);
		hipMemcpy(d_p1,d_p,Nall * sizeof(T), hipMemcpyDeviceToDevice);
	}
	//iterate until convergence
	Scalar res = 0;
	Int iterations = 0;

	/* **************************
	 * Iterative solvers
	 * *************************/
	while(iterations < Controls::max_iterations) {
        /*counter*/
		iterations++;

		/*select solver*/
		if(Controls::Solver == Controls::SOR) {
			iterations++;
			cudaJacobi <<< nBlocks, nThreads >>> (d_rows,d_cols,d_an,d_cF,d_AP,d_Su,d_r,N,Controls::SOR_omega);
			cudaJacobi <<< nBlocks, nThreads >>> (d_rows,d_cols,d_an,d_AP,d_cF,d_Su,d_r,N,Controls::SOR_omega);
		} else if(M.flags & M.SYMMETRIC) {
			/*conjugate gradient   : from wiki*/
			cudaMul <<< nBlocks, nThreads >>> (d_rows,d_cols,d_an,N,d_p,d_AP);
			oo_rr = cudaTdot(d_p,d_AP,d_sum,sum,nBlocks32,N);
			alpha = sdiv(o_rr , oo_rr);
			cudaTaxpy <<< nBlocks, nThreads >>> (N,alpha,d_p,d_cF,d_cF);
			cudaTaxpy <<< nBlocks, nThreads >>> (N,-alpha,d_AP,d_r,d_r);
			oo_rr = o_rr;
			cudaTxmy <<< nBlocks, nThreads >>> (N,d_r,d_pC,d_AP);
			o_rr = cudaTdot(d_r,d_AP,d_sum,sum,nBlocks32,N);
			beta = sdiv(o_rr , oo_rr);
			cudaTaxpy <<< nBlocks, nThreads >>> (N,beta,d_p,d_AP,d_p);
			/*end*/
		} else {
			/* biconjugate gradient : from wiki */
			cudaMul <<< nBlocks, nThreads >>> (d_rows,d_cols,d_an,N,d_p,d_AP);
			cudaMul <<< nBlocks, nThreads >>> (d_rows,d_cols,d_anT,N,d_p1,d_AP1);
			oo_rr = cudaTdot(d_p1,d_AP,d_sum,sum,nBlocks32,N);
			alpha = sdiv(o_rr , oo_rr);
			cudaTaxpy <<< nBlocks, nThreads >>> (N,alpha,d_p,d_cF,d_cF);
			cudaTaxpy <<< nBlocks, nThreads >>> (N,-alpha,d_AP,d_r,d_r);
			cudaTaxpy <<< nBlocks, nThreads >>> (N,-alpha,d_AP1,d_r1,d_r1);
			oo_rr = o_rr;
			cudaTxmy <<< nBlocks, nThreads >>> (N,d_r,d_pC,d_AP);
			cudaTxmy <<< nBlocks, nThreads >>> (N,d_r1,d_pC,d_AP1);
			o_rr = cudaTdot(d_r1,d_AP,d_sum,sum,nBlocks32,N);
			beta = sdiv(o_rr , oo_rr);
			cudaTaxpy <<< nBlocks, nThreads >>> (N,beta,d_p,d_AP,d_p);
			cudaTaxpy <<< nBlocks, nThreads >>> (N,beta,d_p1,d_AP1,d_p1);
		}

		/* *********************************************
		* calculate norm of residual & check convergence
		* **********************************************/
		local_res[0] = cudaTdot(d_r,d_r,d_sum,sum,nBlocks32,N);
		local_res[1] = cudaTdot(d_cF,d_cF,d_sum,sum,nBlocks32,N);
		res = sqrt(mag(local_res[0]) / mag(local_res[1]));
		
		/*check convergence*/
		if(res <= Controls::tolerance)
			break;
	}

	/*****************************
	 *  Copy result back to cpu
	 *****************************/
    //copy result
	hipMemcpy(&((*M.cF)[0]), d_cF, N * sizeof(T), hipMemcpyDeviceToHost);

	//update boundary conditons
	updateExplicitBCs(*M.cF);

	//info
	MP::print("Iterations %d Residue: %.5e\n",iterations,res);
	/*********************************
	 * free device memory
	 ********************************/
	{
		hipFree(d_rows);
		hipFree(d_cols);
		hipFree(d_an);
		hipFree(d_cF);
		hipFree(d_Su);

		hipFree(d_r);
		hipFree(d_sum);
		free(sum);

		if(Controls::Solver == Controls::SOR) {
			hipFree(d_AP);
		} else if(Controls::Solver == Controls::PCG) {
			hipFree(d_p);
			hipFree(d_AP);
			hipFree(d_pC);
			if(!(M.flags & M.SYMMETRIC)) {
				hipFree(d_r1);
				hipFree(d_p1);
				hipFree(d_AP1);
				hipFree(d_anT);
			}
		}
	}
	/******************
	 *    END
	 ******************/
}

/***************************
 * Explicit instantiations
 ***************************
void Solve(const MeshMatrix<Scalar>& A) {
	applyImplicitBCs(A);
	SolveT(A);
}
void Solve(const MeshMatrix<Vector>& A) {
	applyImplicitBCs(A);
	SolveT(A);
}
void Solve(const MeshMatrix<STensor>& A) {
	applyImplicitBCs(A);
	SolveT(A);
}
void Solve(const MeshMatrix<Tensor>& A) {
	applyImplicitBCs(A);
	SolveT(A);
}
/* ********************
 *        End
 * ********************/
